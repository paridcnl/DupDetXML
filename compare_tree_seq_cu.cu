#include "hip/hip_runtime.h"
#include "b.h"
#include "inputfile.h"
#include "structdef.h"
#include "structures.h"
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <pthread.h>
#include <unistd.h>
#include <stdlib.h>
#include "time.h"

#define DEBUG 0

//final querying output
typedef struct t_q_res{
unsigned match_count;
}query_res;

static query_res QueryResult[MAX_NUM_QUERY];

extern "C" int output_query_matches(void);
//all the cuda related functions
#include "header.h"

//forward declarations
void print_input();
void verify_result( MyMatch * match, unsigned int Q );

extern "C" int kernel_wrapper(int t_num, int document_count, doc_metadata * document, short * rp_nodelist, int curr_rp_node_ptr );

#define BLOCK_SZ 256
#define BLOCK_SZ 1024
#define MAX_TREE_SEQ_SIZE 1200

__global__ void GPU_kernel( int N, doc_metadata * document, short * rp_nodelist, int * match_state, int threadsPerBlock, int curr_rp_node_ptr )
{	
	int RefTreeID;
	int CandTreeID;

	int start_idx1, end_idx1, start_idx2, end_idx2;
        int ref_tree_size;
        int cand_tree_size;

        unsigned short ref_tree[MAX_TREE_SEQ_SIZE];
        unsigned short cand_tree[MAX_TREE_SEQ_SIZE];
        unsigned short out_XOR[MAX_TREE_SEQ_SIZE];

        int num_iter;
        int max_iter = 1;
        //int max_iter = 4;
	//int max_iter = 10;
        int i, j, temp;
        int new_ref_tree_size = 0;
        int temp_idx;

	int NumBlocks = N/BLOCK_SZ;

	//if ( blockIdx.y < N )
	if ( blockIdx.x <= N )
	{ 
	//if ( threadIdx.x < threadsPerBlock )
	if ( threadIdx.x <=  N )
	{
		//RefTreeID = blockIdx.y + 1;
		RefTreeID = blockIdx.x + 1;
        	start_idx1 = document[RefTreeID].start;
                end_idx1 = document[RefTreeID].end;
	        ref_tree_size = end_idx1 - start_idx1 + 1;

		max_iter = ref_tree_size;

	       	for (i=0; i < ref_tree_size; i++)	{ ref_tree[i] = rp_nodelist[start_idx1 + i];   }
                //init out_XOR tree
                for (i=0; i < ref_tree_size; i++)	{    out_XOR[i] = 0;    }

		//CandTreeID = (blockIdx.x)*(BLOCK_SZ) + threadIdx.x + 1;
		//CandTreeID = threadIdx.x + 1 + 259;
		CandTreeID = threadIdx.x + 1;

		if ( RefTreeID != CandTreeID )
		{
	                start_idx2 = document[CandTreeID].start;
        	        end_idx2 = document[CandTreeID].end;
                	cand_tree_size = end_idx2 - start_idx2 + 1;

		        for (i=0; i < cand_tree_size; i++)	{ cand_tree[i] = rp_nodelist[start_idx2 + i]; }
		        new_ref_tree_size = ref_tree_size;

			num_iter = 0;

	        	while( num_iter++ < max_iter ){
				
                		for ( j=0; j < new_ref_tree_size; j++ ){
                        		out_XOR[j] = ( ref_tree[j] )^( cand_tree[j] );
                		}
	                	temp = 0;

        		        for ( i=0; i < new_ref_tree_size; i++) {
	                        	if ( out_XOR[i] != 0 ) { //then retain the element
        	                        	ref_tree[i] = ref_tree[i];
                	                	cand_tree[i] = cand_tree[i];
                        	        	temp++;
                        		}
                        		else { //search for next non-zero element
		                                temp_idx = i;
        		                        while ( ++temp_idx < new_ref_tree_size ) {
                		                        if ( out_XOR[temp_idx] != 0 )
                        	                        	break;
                        		        }
	                                	if ( temp_idx == new_ref_tree_size )
        	                                	break;
                	                	else {
	                                        	ref_tree[i] = ref_tree[temp_idx];
        	                                	cand_tree[i] = cand_tree[temp_idx];
                	                        	temp++;
                        	        	}
                        		}
                		}	

	                	for (i=0; i < ref_tree_size; i++)	{      out_XOR[i] = 0;         }

        	        	new_ref_tree_size = temp;	
				
                		if ( !new_ref_tree_size ){
					match_state[RefTreeID ] = CandTreeID;			
		                        break;
        	        	}
	        	}//end of WHILE loop
		}//end of IF
	}//end of thread	
	}//end of block
	//}//end of grid

	return;
}//end of kernel

static int no_device_yet = 1; 
static int grab_gpu_device_attempt = 0;

int kernel_wrapper(int t_num, int document_count, doc_metadata * document, short * rp_nodelist, int curr_rp_node_ptr )
{
	unsigned int r;	int i;
	unsigned int r1;
	int num = t_num;
	//int match_state[520];
	int match_state[1879*2];

	//init match_state
	//for (i=0; i < 520; i++)	match_state[i] = -1;
	for (i=0; i < 1879*2; i++)	match_state[i] = -1;

       	srand( time(NULL) );	r = rand() % 10000;   	usleep ( r );         fprintf(stdout, "\nsleeping for r=%d second\n", r/1000000);
	
        no_device_yet = grab_gpu_device(); 

        while ( no_device_yet == 1 ) {	
		r = rand();	        r1 = ( 2000000 + ( r % 4000000 ));
	        fprintf(stdout, "\nr1=%d", r1); fprintf(stdout, "\nsleeping for r1=%d second", r1/1000000);
		usleep ( r1 );
	        //task_outcome_flag = kernel_wrapper( t_num, query_info1, query_array, rp_nodelist, tag_index_matrix, tag_id_ctr, curr_rp_node_ptr );
        	no_device_yet = grab_gpu_device(); 	        grab_gpu_device_attempt++;
        	if ( grab_gpu_device_attempt >= 4 ) 		{  	return 1;	}
        }

	//dummy code to initiate driver initialization
	if (num == 1)	{
	        MyMatch * d_a1;        	int size1 = sizeof( MyMatch ); 		
		hipMalloc( (void **)&d_a1, size1*1 );	fprintf(stdout, "\ndummy call");	return 0;
	}

	doc_metadata * d_document;	short * d_rp_nodelist;	int * d_match_state;

        srand( time(NULL) );         r = rand() % 10000;         usleep ( r );        no_device_yet = grab_gpu_device();

        while ( no_device_yet == 1 )
        {       r = rand();	r1 = 0 ;
                if (DEBUG) fprintf(stdout, "\n sleeping for r1=%d second", r1/1000000);
                usleep ( r1 );                 no_device_yet = grab_gpu_device();
                grab_gpu_device_attempt++;
                if (DEBUG) fprintf(stdout, "\n grab_gpu_device_attempt=%d", grab_gpu_device_attempt );
                if ( grab_gpu_device_attempt >= 4 )	return 1;
        }

	clock_t start1=clock();		printf("\n start:%lu\n", start1 );

	int N = document_count;

	hipMalloc((void**)&d_document, sizeof(doc_metadata)*N );
	hipMalloc((void**)&d_rp_nodelist, sizeof(short)*curr_rp_node_ptr );
	hipMalloc((void**)&d_match_state, sizeof(int)*N );

	//Copy vectors from host memory to device memory
	hipMemcpy(d_document, document, sizeof(doc_metadata)*N, hipMemcpyHostToDevice );
	hipMemcpy(d_rp_nodelist, rp_nodelist, sizeof(short)*curr_rp_node_ptr , hipMemcpyHostToDevice );
	hipMemcpy(d_match_state, match_state, sizeof(int)*N, hipMemcpyHostToDevice );

	printf("\n data transfer complete");

	// Invoke kernel
	int threadsPerBlock = BLOCK_SZ; 

	int blocksPerGrid; 	
	blocksPerGrid = N/threadsPerBlock;
	//dim3 blocksGrid( N, blocksPerGrid );
	int blocksGrid = N;
	int numGrids = N;	

	fprintf(stdout,"\n blocksPerGrid=%d \t threadsPerBlock=%d \t N=%d \t document_count=%d", blocksPerGrid, threadsPerBlock, N, document_count );
	get_cuda_error("\n before kernel launch:");

	clock_t start2=clock();

	//GPU_kernel<<< blocksGrid, threadsPerBlock >>>(N, d_document, d_rp_nodelist, d_match_state, numGrids, blocksPerGrid, threadsPerBlock);
	GPU_kernel<<< blocksGrid, threadsPerBlock >>>(N, d_document, d_rp_nodelist, d_match_state, threadsPerBlock, curr_rp_node_ptr );

	get_cuda_error("\n after kernel launch:");
	/* #ifdef _DEBUG cutilSafeCall( cutilDeviceSynchronize() ); #endif 
	*/

	hipMemcpy(match_state, d_match_state, sizeof(int)*N, hipMemcpyDeviceToHost );
	get_cuda_error("\n after hipMemcpy device 2 host");
	printf("\n printing result \n");
	// Verify result

	int sum=0;
	//int prev_qid = 0; 	int curr_qid = 0;
	
	//for ( i = 0; i < NUM_QUERY_NODES; i=i+1 ) 
	for ( i = 1; i <= N; i++ ) 
	{	
		//if ( ( match_state[i] > 0 )) 
		{
			printf("\n i = %d \t match_state = %d", i, match_state[i] );		    	
        		sum++;
		}
	}
	
	printf("\n total matches: %d", sum);

	clock_t end=clock();
	printf("\n end:%lu \n", end );
	printf("\n GPU kernel: elapsed time (end-start1) =  %4.4f \n", (float)(end-start1)/CLOCKS_PER_SEC  );
	printf("\n GPU kernel: elapsed time (end-start2) =  %4.4f \n", (float)(end-start2)/CLOCKS_PER_SEC  );

	hipFree( d_document );
	hipFree( d_rp_nodelist );
	hipFree( d_match_state );
	hipDeviceReset();

	return 0;
}
// Functions
void CleanupResources(void);
void RandomInit(float*, int);
void ParseArguments(int, char**);



